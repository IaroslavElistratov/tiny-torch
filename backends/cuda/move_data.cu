
// comment:
// keep in mind that there's an asymmetry between copy_to_device (which
// actually overwrites t->data), and copy_to_host (which returns a new tensor)

// this wrapper macro is needed to pass __FILE__, __LINE__ to the fn,
// because macro (unlike fn) gets expanded at the call sight
#define checkCudaErrors(ans) {_checkCudaErrors((ans), __FILE__, __LINE__);}
inline void _checkCudaErrors(hipError_t err, const char *file, int line) {
    if (err != hipSuccess){
        fprintf(stderr,"[checkCudaErrors] error: %s %s %d\n", hipGetErrorString(err), file, line);
        exit(1);
    }
}

// #define checkCudaErrors(msg) \
//     do{\
//         hipError_t __err = hipGetLastError(); \
//         if (__err != hipSuccess) { \
//             fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
//                 msg, hipGetErrorString(__err), \
//                 __FILE__, __LINE__); \
//             fprintf(stderr, "*** FAILED - ABORTING ***\n"); \
//             exit(1); \
//         } \
//     } while (0)


void set_backend_cuda(void);

void copy_to_cuda(tensor* t){
    if (DATA_COPY_DEBUG) printf("copy_to_cuda\n");
    if (t->device == CUDA){
        return;
    }
    if (t->device != CPU){
        printf("[copy_to_cuda] expected device CPU\n");
        exit(1);
    }

    // question-now: should I do contigify before hipMemcpy?
    assert_contiguous(t);

    // not needed for copying data itself,
    // but need it bc this fn can be called
    // from inside a tensor constructor --
    // in which case it will set this member

    float* t_device;
    int size = t->size * sizeof(float);
    checkCudaErrors(hipMalloc((void**)&t_device, size));
    checkCudaErrors(hipMemcpy(t_device, t->data, size, hipMemcpyHostToDevice));

    // should free (even assuming that "t" is in the GC list) bc the GC stores pointers
    // to Tensors, not to their underlying data -- so when data got changed the GC
    // does not see this
    free(t->data);
    t->data = t_device;
    t->device = CUDA;
}


tensor* copy_from_cuda(tensor* t) {
    if (DATA_COPY_DEBUG) printf("copy_from_cuda\n");
    if (t->device == CPU){
        return t;
    }
    if (t->device != CUDA){
        printf("[copy_from_cuda] expected device CUDA\n");
        exit(1);
    }
    assert_contiguous(t);

    // todo: can just define a macro for print to call 4 lines below and then call the orignal print2d (no need for cuda_print_2d)
    hipDeviceSynchronize();
    int size = t->size * sizeof(float);
    float* host_data = (float*)checkMallocErrors(malloc(size));
    checkCudaErrors(hipMemcpy(host_data, t->data, size, hipMemcpyDeviceToHost));
    // avoids TensorLike returning a cuda tensor (TensorLike->TensorNd->COPY_TO_DEVICE->copy_to_cuda)

    // no need to set backend to CPU (before constructing the tensor)
    // and then back, because TensorLikeNoData (unlike TensorLike) does
    // NOT invoke COPY_FROM_DEVICE
    tensor* t_copy = TensorLikeNoData(t);
    t_copy->data = host_data;
    t_copy->device = CPU;

    // note: no need to free "t", assuming it was created with tensor constructor then it's already traced by the GC array
    add_to_gc(t_copy);

    return t_copy;
}

void set_backend_cuda(void){
    COPY_TO_DEVICE = copy_to_cuda;
    COPY_FROM_DEVICE = copy_from_cuda;
}
