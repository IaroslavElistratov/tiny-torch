
// comment:
// keep in mind that there's an asymmetry between copy_to_device (which
// actually overwrites t->data), and copy_to_host (which returns a new tensor)

inline void checkCudaErrors(hipError_t err) {
    // todo: exit from program everywhere in case of error
    if (err != hipSuccess){
        printf("[cuda malloc/memcopy] error: %s\n",  hipGetErrorString(err));
        exit(1);
    }
}

void set_backend_cuda(void);

void copy_to_cuda(tensor* t){
    if (DATA_COPY_DEBUG) printf("copy_to_cuda\n");
    if (t->device == CUDA){
        return;
    }
    if (t->device != CPU){
        printf("[copy_to_cuda] expected device CPU\n");
        exit(1);
    }

    // question-now: should I do contigify before hipMemcpy?
    assert_contiguous(t);

    // not needed for copying data itself,
    // but need it bc this fn can be called
    // from inside a tensor constructor --
    // in which case it will set this member

    float* t_device;
    int size = t->size * sizeof(float);
    checkCudaErrors(hipMalloc((void**)&t_device, size));
    checkCudaErrors(hipMemcpy(t_device, t->data, size, hipMemcpyHostToDevice));
    // todo: free cpu t->data (currently memory leak)
    t->data = t_device;
    t->device = CUDA;
}


tensor* copy_from_cuda(tensor* t) {
    if (DATA_COPY_DEBUG) printf("copy_from_cuda\n");
    if (t->device == CPU){
        return t;
    }
    if (t->device != CUDA){
        printf("[copy_from_cuda] expected device CUDA\n");
        exit(1);
    }
    assert_contiguous(t);

    // todo: can just define a macro for print to call 4 lines below and then call the orignal print2d (no need for cuda_print_2d)
    hipDeviceSynchronize();
    int size = t->size * sizeof(float);
    float* host_data = (float*)malloc(size);
    checkCudaErrors(hipMemcpy(host_data, t->data, size, hipMemcpyDeviceToHost));
    // avoids TensorLike returning a cuda tensor (TensorLike->TensorNd->COPY_TO_DEVICE->copy_to_cuda)

    // no need to set backend to CPU (before constructing the tensor)
    // and then back, because TensorLikeNoData (unlike TensorLike) does
    // NOT invoke COPY_FROM_DEVICE
    tensor* t_copy = TensorLikeNoData(t);
    t_copy->data = host_data;
    t_copy->device=CPU;
    // todo: free t, currently memory leak
    return t_copy;
}

void set_backend_cuda(void){
    COPY_TO_DEVICE = copy_to_cuda;
    COPY_FROM_DEVICE = copy_from_cuda;
}
