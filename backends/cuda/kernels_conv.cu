#include "hip/hip_runtime.h"
#define NUM_THREADS 32
#define CUDA_DEBUG true


// input(B, C, H, W) kernel(F, C, HH, WW) = out(B, F, h_out, w_out)
// input(B, C, H, W) kernel(F, C, HH, WW) = out(B, F, h_out, w_out)
__global__ void ConvKernel(float* x, float* kernel, float* out, int F, int H_OUT, int W_OUT, int H, int W, int C, int HH, int WW, bool is_batched){

    // out's idxs
    int curr_height = blockIdx.x * blockDim.x + threadIdx.x;
    int curr_width = blockIdx.y * blockDim.y + threadIdx.y;
    int b = blockIdx.z * is_batched;

    if (curr_height<H_OUT && curr_width<W_OUT){

        // e.g. 5/2=2 (with int division)

        // todo-now: bug was here.
        // int h_start = curr_height - (HH/2);
        // int w_start = curr_width - (WW/2);

        int h_start = curr_height - ((HH-1)/2);
        int w_start = curr_width - ((WW-1)/2);

        for (int f=0; f<F; f++){

            float curr_out = 0.0;

            // iterate over all elements where kernel overlays on the input

            // kernel's idxs
            for (int h=0; h<HH; h++){
                for (int w=0; w<WW; w++){

                    // x's idxs
                    int x_h = h_start + h;
                    int x_w = w_start + w;

                    // handles ghost cells (same semantics as padding with zeros)
                    if (x_h>-1 && x_h<H && x_w>-1 && x_w<W){

                            // for each input channel
                            for (int c=0; c<C; c++)
                                curr_out += x[b*C*H*W + c*H*W + x_h*W + x_w] * kernel[f*C*HH*WW + c*HH*WW + h*WW + w];

                    }
                }
            }

            out[b*F*H_OUT*W_OUT + f*H_OUT*W_OUT + curr_height*W_OUT + curr_width] = curr_out;

        }
    }

}


// input(C, H, W) kernel(F, C, HH, WW) = out(F, h_out, w_out)
tensor* conv_k(tensor* input, tensor* kernel){
    if (CUDA_DEBUG) printf("[conv_k]\n");
    // unary_batched_input_checks(input);

    int C = input->shape[0], H = input->shape[1], W = input->shape[2];
    int F = kernel->shape[0], HH = kernel->shape[2], WW = kernel->shape[3];

    if (input->num_dims!=3 || kernel->num_dims!=4){
        printf("[cuda conv_k] expected 3-d input and 4-d kernel\n");
        exit(1);
    }
    if (WW!=HH){
        printf("[cuda conv_k] for now conv assumes square kernels\n");
        exit(1);
    }
    if (input->shape[0]!=kernel->shape[1]){
        printf("[cuda conv_k] C-dim doesn't match\n");
        exit(1);
    }

    // todo: change h_out, w_out computation?
    // int h_out = (H - HH + 1) / stride;
    // int w_out = (W - WW + 1) / stride;
    int h_out = H - HH + 1;
    int w_out = W - WW + 1;

    // todo: allocate empty, here and other kenrels
    tensor* out = Tensor(F, h_out, w_out);

    float num_threads = (float)NUM_THREADS;
    // todo-high:
    // One possible design is to just add one more dim for F and remove loop over F from the kernel
    //  Another possible design is to keep the loop over F and (later in batch_conv) add grid.z for B (not F);
    //  With the 2nd approach parallel over B is cleaner in the code bc can have separate block-dim for that B specifically
    //  In the 1st approach need to cram both F and B into grid.z
    dim3 dimGrid(ceil(h_out/num_threads), ceil(w_out/num_threads), 1);
    dim3 dimBlock(num_threads, num_threads, 1);

    if (CUDA_DEBUG){
        printf("[cuda ConvKernel] grid: (%f, %f, 1)\n", ceil(h_out/num_threads), ceil(w_out/num_threads));
        printf("[cuda ConvKernel] block: (%f, %f, 1)\n", num_threads, num_threads);
    }

    ConvKernel<<<dimGrid, dimBlock>>>(input->data, kernel->data, out->data, F, h_out, w_out, H, W, C, HH, WW, false);

    return out;
}


// input (C, H, W) kernel (F, C, HH, WW) = out (F, h_out, w_out)
// input (B, C, H, W) kernel (F, C, HH, WW) = out (B, F, h_out, w_out)
tensor* batched_conv_k(tensor* input, tensor* kernel){
    if (CUDA_DEBUG) printf("[batched_conv_k]\n");
    // unary_batched_input_checks(input);

    int B = input->shape[0], C = input->shape[1], H = input->shape[2], W = input->shape[3];
    int F = kernel->shape[0], HH = kernel->shape[2], WW = kernel->shape[3];

    if (input->num_dims!=4 || kernel->num_dims!=4){
        printf("[cuda batched_conv_k] expected 3-d input and 4-d kernel\n");
        exit(1);
    }
    if (WW!=HH){
        printf("[cuda batched_conv_k] for now conv assumes square kernels\n");
        exit(1);
    }
    if (input->shape[1]!=kernel->shape[1]){
        printf("[cuda batched_conv_k] C-dim doesn't match\n");
        exit(1);
    }

    int h_out = H - HH + 1;
    int w_out = W - WW + 1;

    tensor* out = Tensor(B, F, h_out, w_out);

    float num_threads = (float)NUM_THREADS;
    dim3 dimGrid(ceil(h_out/num_threads), ceil(w_out/num_threads), B);
    dim3 dimBlock(num_threads, num_threads, 1);

    if (CUDA_DEBUG){
        printf("[cuda BatchedConvKernel] grid: (%f, %f, %i)\n", ceil(h_out/num_threads), ceil(w_out/num_threads), B);
        printf("[cuda BatchedConvKernel] block: (%f, %f, 1)\n", num_threads, num_threads);
    }

    ConvKernel<<<dimGrid, dimBlock>>>(input->data, kernel->data, out->data, F, h_out, w_out, H, W, C, HH, WW, true);
    return out;
}
