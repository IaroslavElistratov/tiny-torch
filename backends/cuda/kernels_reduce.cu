#include "hip/hip_runtime.h"
#include "../../nn.h"


// Reduce kernel isn't as straightforward as it seems, because naively it
// requires a lot of atomic operations (poor perf -- worse than CPU), so
// use parallel reduction pattern

// question-now: float atomic add doesn't work for shared memory -- https://github.com/treecode/Bonsai/blob/581fa8e70501ce85660c7eac0d61c0e5c5bece4a/runtime/profiling/derived_atomic_functions.h#L14-L17
// https://github.com/treecode/Bonsai/blob/581fa8e70501ce85660c7eac0d61c0e5c5bece4a/runtime/profiling/derived_atomic_functions.h#L199C1-L209C2
__device__ float atomicMax(float *address, float val){
    int ret = __float_as_int(*address);
    while(val > __int_as_float(ret)){
        int old = ret;
        if((ret = atomicCAS((int *)address, old, __float_as_int(val))) == old)
            break;
    }
    return __int_as_float(ret);
}


// modified _launch_reduction_kernel to save idxs to be used for _bwd
struct value {
    float val;
    int idx;
};

__device__ value max(value a, value b){
    return (a.val > b.val) ? a : b;
}

// for reduce_add, it doesn't make sense to propagate idxs (value->idx)
// through the reduction, but I still do it because want the code inside
// the ReductionKernel to work polymorphically (op_fns, atomic_fns) for
// any reduction op (so that _launch_reduction_kernel can be re-used for:
// reduce_max_k, reduce_sum_k, etc)
__device__ value add(value a, value b){
    // ues -1 as an invalid idx
    return value{a.val + b.val, -1};
}


/*
more generic form to support different reduce funcs with the same kernel

Prior to that tried pointers to the kernel:
    __global__ void ReductionKernel(ReductionFn reduction_fn, AtomicFn atomic_fn, float* input, float* out)

    _launch_reduction_kernel(...)
        ReductionKernel<<<dimGrid, dimBlock>>>(reduction_fn, atomic_fn, input->data, out->data)

    tensor* reduce_max_k(tensor* a)
        return _launch_reduction_kernel(deviceMax, atomicMax, a);

Resulted in err below, likely because the function pointers passed were allocated on CPU (all inputs to the kernel should be allocated on the device)
    CUDA Exception: Warp Invalid PC; Error: Could not read error PC  (dev=0, sm=0, wp=0), error=CUDBG_ERROR_UNKNOWN_FUNCTION(0x3).
    (cuda-gdb)  Program terminated with signal CUDA_EXCEPTION_8, Warp Invalid PC.
*/
typedef value (*OpFn)(value, value);
// typedef float (*AtomicFn)(float *, float);
__device__ OpFn op_fns[2] = {max, add};
// __device__ AtomicFn atomic_fns[2] = {atomicMax, atomicAdd};


// features: convergent (active threads are close to each other), with privatization (shared memory), segmented (multi-block);
// scratch_space (used when ReductionKernel is used to compute reduce_max) to store idxs of max values, needed for the reduce_max_bwd and reduce_max_batched_bwd
__global__ void ReductionKernel(int op_idx, float* input, value* out, int stride_to_next_b, int num_blocks){

    // recover is_batched from stride_to_next_b to avoid passing 2
    // arguments (is_batched, stride_to_next_b) which mostly mean the same thing
    bool is_batched = stride_to_next_b ? true : false;
    // cancels out when not batched
    unsigned int batch = blockIdx.y * is_batched;

    // variable t is the start in the shared array, variable "start" is the start in the input array
    unsigned int t = threadIdx.x;

    // todo-now:
    // Add guards to the kernel so that if the input is smaller than 2*NUM_THREADS, it should not access it's these locations;
    // Even non-batched version of this kernel has this bug; remove exit from the stub

    // *** privatization ***

    // each thread block takes 2*BlockDim.x input elements
    // note: Dynamic allocations happen at kernel invocation
    // __shared__ float partialSum[2*blockDim.x];
    __shared__ value partialSum[2*NUM_THREADS];
    // thus when we multiply the size of each input segment by blockIdx.x
    // of a block, we have the starting location of the segment to be
    // processed by the block
    unsigned int start = batch*stride_to_next_b + 2*blockIdx.x*blockDim.x;

    // each thread loads 2 elements into shared memory
    //  set thread_idx in the shared array to start_idx+thread_idx in the global array

    // initialize value struct, with the index of that value in the input tensor -- needs to be idx of input tensor not local tensor (partial_sum)
    // note: the disadvantage of passing "value structs" around is that for reduce_sum idxs effectively do nothing, but do compute shared memory
    partialSum[t] = {input[start+t], start+t};
    partialSum[t + blockDim.x] = {input[start+t + blockDim.x], start+t + blockDim.x};

    if (CUDA_DEBUG){
        printf("[ReductionKernel] batch: %i\n", batch);
        printf("[ReductionKernel blockIdx.y=%i] partialSum[t=%i] = input[(start + t)=%i];\n", blockIdx.y, t, start+t);
        printf("[ReductionKernel blockIdx.y=%i] partialSum[(t + blockDim.x)=%i] = input[(start+t + blockDim.x)=%i];\n", blockIdx.y, t + blockDim.x, start+t + blockDim.x);
    }

    // *** convergent reduction kernel ***

    // in each step, one of the inputs comes from an increasing distance away
    //  much better for SM's warp utilization to stride/=2 rather than stride/=2
    for (unsigned int stride=blockDim.x; stride>0; stride/=2){
        // after each step, half of the threads are no longer needed;
        // each thread is responsible for an  index location in the partial sum vector ("location of responsibility");
        // tests for wether the thread should be active
        if (t < stride){
            partialSum[t] = op_fns[op_idx](partialSum[t], partialSum[t+stride]);
        }
        __syncthreads();
    }

    // once we finish execution of the for loop for all the threads, we have entire
    // section (2*num_threads in this block) of the input vector reduced to one value,
    // so at the end of the kernel only need to have one thread to write the sum of the
    // thread block into a new vector (to be index with blockIdx.x);
    // In this new vector every element is a partial sum produced by one thread block

    /*
    An alternative way to get final value is instead of using Atomic functions at the end, can
    pass output array of shape num_blocks, then one thread from each block can write the partial
    sum (for the block) into this array indexing that array with its BlockIdx, then have a cpu
    short loop in the stub to sum these per-block partial sums into a final value.

    For now preferred having atomics as it simplifies things by limiting all computations to the
    kernel function (wt needing additional logic in the stub)

    But then extended to propagate value structs though the reduction kernel, and it's not clear
    how to use atomics to reduce two value structs. So ended up pass array of b*num_blocks to back
    to the stub and do the reduction across the blocks there
    todo-now:
    Perform atomic reductions on custom structures -- bc I want to update two adjacent 32-bit items,
    can use a generalized 64-bit atomic operation, treat the entire struct as a single 64-bit value
    (unsigned long long)
    */
    if (t==0){
        out[batch*num_blocks + blockIdx.x] = partialSum[t];
        // atomic_fns[op_idx](&out[batch], partialSum[t].val);
    }
}

tensor* _launch_reduction_kernel(int op_idx, tensor* input, bool is_batched){
    // todo:
    // unary_input_checks(input);

    float fill_value;
    // todo-high: ugly
    if (op_idx==0){
        fill_value = -100.0;
    } else if (op_idx==1){
        fill_value = 0.0;
    } else {
        printf("[cuda reduction_kernel] unsupported op_idx\n");
        exit(1);
    }

    float num_threads = (float)NUM_THREADS;
    int num_blocks, B, stride_to_next_b;
    tensor* out;
    if (!is_batched){
        B = 1;
        // used as (bool is_batch) inside the kernel
        stride_to_next_b = 0;

        out = TensorScalarFill(fill_value);
        // each thread block consumes num_threads*2 inputs
        num_blocks = ceil(input->size/(num_threads*2));
        if (input->size < num_threads*2){
            printf("[temporary] shape err: %i(input->size) < %i(num_threads*2)\n", input->size, (int)num_threads*2);
            exit(1);
        }
    } else {
        B = input->shape[0];
        stride_to_next_b = input->stride[0];

        // defining the second tensor is needed so that you can use Fill (currently I don't
        // conveniently support initializing non 1d tensor and filling it);
        // This isn't necessary bc you overwrite ->data inside the Kernel anyway
        // out = TensorLikeFill(Tensor(B, 1), fill_value);
        out = Tensor(B, 1);

        // In non-batched kernel each thread block consumes num_threads*2 inputs;
        // Additionally, divide by B so that each block doesn't see all the elements
        // (input->size), but instead it only sees elements in a single batch element (b);
        // And because you have B as additional dim of the grid -- these elements
        // will still be covered (but by blocks with different blockIdx.y)
        num_blocks = ceil(input->size/B/(num_threads*2));
        if (input->shape[1] < num_threads*2){
            printf("[temporary] shape err:  %i(input->shape[1]) < %i(num_threads*2)\n", input->shape[1], (int)num_threads*2);
            exit(1);
        }
    }

    dim3 dimGrid(num_blocks, B, 1);
    dim3 dimBlock(num_threads, 1, 1);

    if (CUDA_DEBUG){
        printf("[cuda reduction_kernel] grid: (%i, %i, 1)\n", num_blocks, B);
        printf("[cuda reduction_kernel] block: (%f, 1, 1)\n", num_threads);
        printf("stride_to_next_b: %i\n", stride_to_next_b);
    }

    // copy to cuda, then copy out back to cpu -- maybe a better solution is to launch 2nd kernel (to do the work in reduce_max_bwd)/
    value* out_device; // (B, num_blocks)
    int size = B * num_blocks * sizeof(value);
    checkCudaErrors(hipMalloc((void**)&out_device, size));

    ReductionKernel<<<dimGrid, dimBlock>>>(op_idx, input->data, out_device, stride_to_next_b, num_blocks);

    // copy out back to cpu
    value* out_host = (value*)malloc(size);
    checkCudaErrors(hipMemcpy(out_host, out_device, size, hipMemcpyDeviceToHost));

    // reduction over IDX
    // aggregate values from multiple blocks into a single value
    // use this because not sure how to use max_with_idxs with atomics
    // (to be used inside the kernel)

    // copy to host because below I modify its ->data attribute
    tensor* scratch_space = COPY_FROM_DEVICE(TensorLike(out));
    out = COPY_FROM_DEVICE(out);

    for (int b=0; b<B; b++){
        // blocks for the current b
        value* curr_blocks = out_host + b*num_blocks; // (B, num_blocks) -> (num_blocks, )
        value max = curr_blocks[0]; // (num_blocks, )[0]
        for (int i=1; i<num_blocks; i++){
            value curr = curr_blocks[i]; // (num_blocks, )[i]
            // todo-now: support other reduction, based on op_idx
            // can't re-use max_with_idxs since it's a device function, but this is host code (not inside the kernel)
            max = (max.val > curr.val) ? max : curr;
        }
        out->data[b] = max.val;
        scratch_space->data[b] = (float)max.idx;
    }

    COPY_TO_DEVICE(scratch_space);
    out->scratch_space[0] = scratch_space;

    COPY_TO_DEVICE(out);
    return out;
}   

tensor* reduce_max_k(tensor* a){
    if (CUDA_DEBUG) printf("[reduce_max_k]\n");
    return _launch_reduction_kernel(0, a, false);
}

tensor* reduce_sum_k(tensor* a){
    if (CUDA_DEBUG) printf("[reduce_sum_k]\n");
    return _launch_reduction_kernel(1, a, false);
}

tensor* batched_reduce_max_k(tensor* a){
    if (CUDA_DEBUG) printf("[batched_reduce_max_k]\n");
    return _launch_reduction_kernel(0, a, true);
}

tensor* batched_reduce_sum_k(tensor* a){
    if (CUDA_DEBUG) printf("[batched_reduce_sum_k]\n");
    return _launch_reduction_kernel(1, a, true);
}
