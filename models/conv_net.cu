#include "hip/hip_runtime.h"
#include <iostream> // todo: use C only
using namespace std;


#define DEVICE CUDA
#define NUM_EP 80
#define BATCH_SIZE 128
#define LR 0.001

// #define NUM_EP 150
// #define BATCH_SIZE 2048
// #define LR 0.001

#include "../nn.h"
#include "../tensor.cpp"
#include "../ops.cpp"
#include "../composite_ops.cpp"
#include "../cifar10.cpp"
#include "../print.cpp"
#include "../optim.cpp"
#include "../codegen.cpp"
#include "../serialization.cpp"


/*
https://pytorch.org/tutorials/beginner/blitz/cifar10_tutorial.html
def forward(self, x):
    x = self.conv1(x)   // Conv2d(in_channels=3, out_channels=6, kernel_size=5)
    x = F.relu(x)
    x = self.pool(x)    // MaxPool2d(2, 2)

    x = self.conv2(x)   // Conv2d(in_channels=6, out_channels=16, kernel_size=5)
    x = F.relu(x)
    x = self.pool(x)

    x = torch.flatten(x, 1) // flatten all dimensions except batch

    x = self.fc1(x)     // Linear(in_features=16 * 5 * 5, out_features=120)
    x = F.relu(x)

    x = self.fc2(x)     // Linear(in_features=120, out_features=84)
    x = F.relu(x)

    x = self.fc3(x)     // Linear(in_features=84, out_features=10)
    return x
*/



tensor* forward(tensor* input) {
    int B = input->shape[0];

    tensor* conv1 = batched_conv(input, get_param("kernel1"), get_param("bias_kernel1"));
    set_name(conv1, "conv1");
    tensor* relu1 = relu(conv1);
    set_name(relu1, "relu1");
    tensor* mp1 = batched_maxpool(relu1);
    set_name(mp1, "mp1");

    tensor* conv2 = batched_conv(mp1, get_param("kernel2"), get_param("bias_kernel2"));
    set_name(conv2, "conv2");
    tensor* relu2 = relu(conv2);
    set_name(relu2, "relu2");
    tensor* mp2 = batched_maxpool(relu2);
    set_name(mp2, "mp2");

    tensor* flat = batched_flatten(mp2);
    set_name(flat, "flat");

    tensor* mm1 = matmul(flat, get_param("w1"));
    set_name(mm1, "mm1");
    tensor* lin1 = add(mm1, repeat(get_param("b1"), /*axis = */ 0, /*num_repeats = */ B));
    set_name(lin1, "lin1");
    tensor* relu3 = relu(lin1);
    set_name(relu3, "relu3");

    tensor* mm2 = matmul(relu3, get_param("w2"));
    set_name(mm2, "mm2");
    tensor* lin2 = add(mm2, repeat(get_param("b2"), /*axis = */ 0, /*num_repeats = */ B));
    set_name(lin2, "lin2");
    tensor* relu4 = relu(lin2);
    set_name(relu4, "relu4");

    tensor* mm3 = matmul(relu4, get_param("w3"));
    set_name(mm3, "mm3");
    tensor* lin3 = add(mm3, repeat(get_param("b3"), /*axis = */ 0, /*num_repeats = */ B));
    set_name(lin3, "lin3");
    return lin3;
}


float accuracy(tensor* log_probs, tensor* label){
    // pred idxs
    tensor* probs = exp(log_probs);
    set_name(probs, "probs");
    tensor* pred = batched_reduce_max(probs)->scratch_space[0];
    set_name(pred, "pred");

    pred = COPY_FROM_DEVICE(pred);
    label = COPY_FROM_DEVICE(label);

    // it's not a binary elementwise but same checks
    // assert_binary_elementwise(pred, label);

    int B = pred->shape[0];
    int correct = 0;
    for (int b=0; b<B; b++){
        (pred->data[b] == label->data[b]) ? correct++ : 0;
    }
    float acc = (float)correct / B;
    printf("accuracy: %f (%i/%i)\n", acc, correct, B);
    return acc;
};

tensor* train_step(cifar10* batch, int ep_idx) {

    // *** Net ***
    tensor* logits = forward(batch->input);

    // *** Loss fn ***
    tensor* log_probs = log_softmax(logits);
    tensor* loss = NLL(log_probs, batch->label);

    // *** Zero-out grads ***
    zero_grads();

    // *** Backward ***
    save_num_uses(loss);
    loss->backward(loss);

    if (ep_idx==0){
        // must call generate test BEFORE param update, otherwise asserts
        // on runtime values don't make sense -- bc SGD mutates weights inplace
        if (BATCH_SIZE <= 256){
            generate_test(loss);
        }
        graphviz(loss);
    }

    // *** Optim Step ***
    // note: sgd sensitive to momentum
    // sgd(LR, /* momentum = */ 0.6);
    adam(LR);

    // todo-high: need smt like torch.detach?
    if (accuracy(log_probs, batch->label) > 0.9){
        save_all_params();
    }
    return loss;
}

// todo-low: when define weights (w1, w2, w3) in forward, can use runtime shapes to create these weights.
// But when creating weights in main (in main fn), needed to hardcode these shapes, copying from train_step.
// w1 = Tensor(flat->shape[1], 32);
// w2 = Tensor(relu3->shape[1], 16);
// w3 = Tensor(relu4->shape[1], 10);
int main(void) {
    // random num generator init, must be called once
    // srand(time(NULL));
    srand(123);
    set_backend_device();

    fclose(fopen("./generated/log.txt", "w"));


    // *** Init ***

    int C = 3;
    int F = 6;
    int HH1 = 7;
    int WW1 = 7;

    int HH2 = 6;
    int WW2 = 6;


    tensor* kernel1 = Tensor(F, C, HH1, WW1);
    set_name(kernel1, "kernel1");
    add_param(kernel1);

    tensor* bias_kernel1 = Tensor(F, 1);
    set_name(bias_kernel1, "bias_kernel1");
    add_param(bias_kernel1);


    tensor* kernel2 = Tensor(F, F, HH2, WW2);
    set_name(kernel2, "kernel2");
    add_param(kernel2);

    tensor* bias_kernel2 = Tensor(F, 1);
    set_name(bias_kernel2, "bias_kernel2");
    add_param(bias_kernel2);


    tensor* w1 = Tensor(96, 64);
    set_name(w1, "w1");
    add_param(w1);

    tensor* b1 = Tensor(1, 64);
    set_name(b1, "b1");
    add_param(b1);


    tensor* w2 = Tensor(64, 32);
    set_name(w2, "w2");
    add_param(w2);

    tensor* b2 = Tensor(1, 32);
    set_name(b2, "b2");
    add_param(b2);


    tensor* w3 = Tensor(32, 10);
    set_name(w3, "w3");
    add_param(w3);

    tensor* b3 = Tensor(1, 10);
    set_name(b3, "b3");
    add_param(b3);


    // todo-low: change add_param to accept array of all prams "add_param({kernel1, bias_kernel1, kernel2, bias_kernel2, w1, w2, w3})"?


    // todo: somehow if having prints in-between initialization of later weights print produces differently
    // initialized later tensors (via print->copy_from_cuda) even though the constructor called from
    // copy_from_cuda does not explicitly advacne the RNG state (does not call GetRandomFloat).
    // question-now: is it bc of "copy_from_cuda -> malloc(size)" ?
    // So moved prints and "get_cifar10" after initializing the tensors -- this way tensors will get initialized to
    // the same values regardless of wether there are prints or not

    // load_all_params();

    cifar10* dataset = get_cifar10();
    int gc_until = GC_IDX;

    // *** Train ***
    print_num_params();

    for (int ep_idx=0; ep_idx<NUM_EP; ep_idx++) {
        cifar10* batch = sample_batch(dataset, BATCH_SIZE, /* is_random = */ false);
        // passes loss sanity check -- 10 classes, if model is random (predicting each cls equally)
        // log(0.1) = -2.3
        tensor* loss = train_step(batch, ep_idx);
        printf("ep: %i; loss: %f;\n\n", ep_idx, COPY_FROM_DEVICE(loss)->data[0]);

        free_all_tensors(gc_until);
    }

    // lprint(params.w3->grad);
    // lprint(params.w2->grad);
    // lprint(params.w1->grad);
    // lprint(params.kernel2->grad);
    // lprint(params.bias_kernel2->grad);
    // lprint(params.kernel1->grad);
    // lprint(params.bias_kernel1->grad);

    // lprint(kernel1);
    // lprint(kernel2);
    // lprint(w1);
    // lprint(w2);
    // lprint(w3);

    hipDeviceReset();
    return 0;
}
