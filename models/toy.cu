#include "hip/hip_runtime.h"
#include <iostream> // todo: use C only
using namespace std;

#define DEVICE CUDA

#include "../nn.h"
#include "../tensor.cpp"
#include "../ops.cpp"
#include "../utils.cpp" // graphviz
#include "../print.cpp"

// todo-now:
// for cuda, use same operation abstractions (as for cpu), but make these abstractions call host stubs for cuda kernels (instead of my kernles for cpu) -- this will preserve my graph building functionality
//   - re-use this from ops.cpp: use same names for kernels, just make them refer to different impls (cpu, cuda) depending on wether device is CUDA or not -- this will reduce code duplication needed to copy paste ops

// todo-now:
// and "matmul_bwd", "batched_matmul_bwd", "div_bwd", "pow_bwd", "reduce_sum_bwd" can also be re-used!
// Seems possible to overwrite all tensor constructors (from CPU to CUDA) when DEVICE macro is set to CUDA, and re-use _bwd[s] ?
//  No need to overwrite constructors, just move selection of what subclass to call (cpu or cuda) to the constructor fn itself

// todo-now:
// Need to abstract away loop over individual elements and then can re-use most of the fwd kernels


// // test_reduce
// int main() {
//     srand(123);
//     set_backend_device();

//     int B = 2, H = 16;

//     // todo-high: currently don't support num_elements lower than NUM_THREADS*2 -- add guards to the kernel
//     tensor* x = Tensor(B, H);
//     set_name(x, "x"); print(x);

//     // tensor* out = reduce_sum(x);
//     tensor* out = reduce_max(x);
//     set_name(out, "out"); print(out);

//     out->backward(out);
//     return 0;
// }



// test_batched_reduce
int main() {
    srand(123);
    set_backend_device();

    int B = 2, H = 32;

    // todo-high: currently don't support 2nd dim lower than NUM_THREADS*2 -- add guards to the kernel
    tensor* x = Tensor(B, H);
    set_name(x, "x"); print(x);

    // tensor* out = batched_reduce_sum(x);
    tensor* out = batched_reduce_max(x);
    set_name(out, "out"); print(out);

    // out->backward(out);
    return 0;
}

// int test_backends(){
//     srand(123);

//     int N = 16, M = 8;

//     set_backend_device();
//     tensor* cuda_t = Tensor(N, M);
//     set_name(cuda_t, "cuda_t"); print(cuda_t);

//     set_backend_cpu();
//     tensor* cpu_tens = Tensor(N, M);
//     set_name(cpu_tens, "cpu_tens"); print(cpu_tens);
//     return 0;
// }


// int test_bmm() {
//     srand(123);

//     int B=3, N = 2, M = 8, D = 4;

//     set_backend_device();

//     tensor* x = Tensor(B, N, M);
//     set_name(x, "x"); print(x);

//     tensor* w1 = Tensor(B, M, D);
//     set_name(w1, "w1"); print(w1);

//     tensor* out = batched_matmul(x, w1);
//     set_name(out, "out"); print(out);

//     out->backward(out);
//     return 0;
// }



// int test_conv() {
//     srand(123);

//     int H = 4, W = 4, C = 3, F = 5, K = 2;

//     set_backend_device();

//     tensor* x = Tensor(C, H, W);
//     set_name(x, "x"); print(x);

//     tensor* kernels = Tensor(F, C, K, K);
//     set_name(kernels, "kernels"); print(kernels);

//     tensor* out = conv(x, kernels);
//     set_name(out, "out"); print(out);

//     out->backward(out);
//     return 0;
// }


// int test_batched_conv() {
//     srand(123);

//     // int B = 3000, H = 128, W = 128, C = 3, F = 5, K = 2;
//     int B = 2, H = 4, W = 4, C = 3, F = 5, K = 2;

//     set_backend_device();

//     tensor* x = Tensor(B, C, H, W);
//     set_name(x, "x"); print(x);

//     tensor* kernels = Tensor(F, C, K, K);
//     set_name(kernels, "kernels"); print(kernels);

//     tensor* out = batched_conv(x, kernels);
//     set_name(out, "out"); print(out);

//     out->backward(out);
//     return 0;
// }


// int test_pool() {
//     srand(123);

//     int H = 4, W = 4, C = 3, K = 2;

//     set_backend_device();

//     tensor* x = Tensor(C, H, W);
//     set_name(x, "x"); print(x);

//     tensor* out = maxpool(x);
//     set_name(out, "out"); print(out);

//     out->backward(out);
//     return 0;
// }


// int test_batched_pool() {
//     srand(123);

//     int B = 2, H = 4, W = 4, C = 3, K = 2;

//     set_backend_device();

//     tensor* x = Tensor(B, C, H, W);
//     set_name(x, "x"); print(x);

//     tensor* out = batched_maxpool(x);
//     set_name(out, "out"); print(out);

//     out->backward(out);
//     return 0;
// }



int test_simple_ops() {
    srand(123);

    int N = 2, M = 8, D = 4;

    // by default set to DEVICE backend
    set_backend_device();

    tensor* x = Tensor(N, M);
    set_name(x, "x"); print(x);

    tensor* w1 = Tensor(M, D);
    set_name(w1, "w1"); print(w1);

    tensor* mm = matmul(x, w1);     // (N, D)
    print(mm);

    tensor* tr = transpose(mm);     // (D, N)
    print(tr);

    tensor* w2 = TensorLike(tr);    // (D, N)
    set_name(w2, "w2"); print(w2);
    tensor* ad = add(tr, w2);       // (D, N)
    print(ad);

    tensor* w3 = TensorLike(ad);    // (D, N)
    set_name(w3, "w3"); print(w3);
    // todo: hangs when replacing "ad" w "tr" below
    tensor* su = sub(ad, w3);       // (D, N)
    print(su);

    tensor* w4 = TensorLike(su);    // (D, N)
    set_name(w4, "w4"); print(w4);
    tensor* di = div(su, w4);       // (D, N)
    print(di);

    tensor* pw = pow(di, 2);
    tensor* ng = neg(pw);
    tensor* ex = exp(ng);
    tensor* out = log(ex);
    set_name(out, "out"); print(out);

    out->backward(out);
    graphviz(out);

    return 0;
}
