#include "hip/hip_runtime.h"
#include <iostream> // todo: use C only
using namespace std;

#define DEVICE CUDA

#include "../nn.h"
#include "../tensor.cpp"
#include "../ops.cpp"
#include "../utils.cpp" // graphviz
#include "../print.cpp"

// todo-now:
// for cuda, use same operation abstractions (as for cpu), but make these abstractions call host stubs for cuda kernels (instead of my kernles for cpu) -- this will preserve my graph building functionality
//   - re-use this from ops.cpp: use same names for kernels, just make them refer to different impls (cpu, cuda) depending on wether device is CUDA or not -- this will reduce code duplication needed to copy paste ops

// todo-now:
// and "matmul_bwd", "batched_matmul_bwd", "div_bwd", "pow_bwd", "reduce_sum_bwd" can also be re-used!
// Seems possible to overwrite all tensor constructors (from CPU to CUDA) when DEVICE macro is set to CUDA, and re-use _bwd[s] ?
//  No need to overwrite constructors, just move selection of what subclass to call (cpu or cuda) to the constructor fn itself

// todo-now:
// Need to abstract away loop over individual elements and then can re-use most of the fwd kernels

void test_backends(void){
    srand(123);

    int N = 16, M = 8;

    set_backend_device();
    tensor* cuda_t = Tensor(N, M);
    set_name(cuda_t, "cuda_t"); print(cuda_t);

    set_backend_cpu();
    tensor* cpu_tens = Tensor(N, M);
    set_name(cpu_tens, "cpu_tens"); print(cpu_tens);
}


int test_bmm() {
    srand(123);

    int B=3, N = 2, M = 8, D = 4;

    set_backend_device();

    tensor* x = Tensor(B, N, M);
    set_name(x, "x"); print(x);

    tensor* w1 = Tensor(B, M, D);
    set_name(w1, "w1"); print(w1);

    tensor* out = batched_matmul(x, w1);
    set_name(out, "out"); print(out);

    out->backward(out);
    return 0;
}



// // test_conv
int main() {
    srand(123);

    int H = 4, W = 4, C = 3, F = 5, K = 2;

    set_backend_device();

    tensor* x = Tensor(C, H, W);
    set_name(x, "x"); print(x);

    tensor* kernels = Tensor(F, C, K, K);
    set_name(kernels, "kernels"); print(kernels);

    tensor* out = conv(x, kernels);
    set_name(out, "out"); print(out);

    // out->backward(out);
}



// int test_batched_conv() {
//     srand(123);

//     // int B = 3000, H = 128, W = 128, C = 3, F = 5, K = 2;
//     int B = 2, H = 4, W = 4, C = 3, F = 5, K = 2;

//     set_backend_device();

//     tensor* x = Tensor(B, C, H, W);
//     set_name(x, "x"); print(x);

//     tensor* kernels = Tensor(F, C, K, K);
//     set_name(kernels, "kernels"); print(kernels);

//     tensor* out = batched_conv(x, kernels);
//     set_name(out, "out"); print(out);

//     // out->backward(out);
// }




int test_simple_ops() {
    srand(123);

    int N = 2, M = 8, D = 4;

    // by default set to DEVICE backend
    set_backend_device();

    tensor* x = Tensor(N, M);
    set_name(x, "x"); print(x);

    tensor* w1 = Tensor(M, D);
    set_name(w1, "w1"); print(w1);

    tensor* mm = matmul(x, w1);     // (N, D)
    print(mm);

    tensor* tr = transpose(mm);     // (D, N)
    print(tr);

    tensor* w2 = TensorLike(tr);    // (D, N)
    set_name(w2, "w2"); print(w2);
    tensor* ad = add(tr, w2);       // (D, N)
    print(ad);

    tensor* w3 = TensorLike(ad);    // (D, N)
    set_name(w3, "w3"); print(w3);
    // todo: hangs when replacing "ad" w "tr" below
    tensor* su = sub(ad, w3);       // (D, N)
    print(su);

    tensor* w4 = TensorLike(su);    // (D, N)
    set_name(w4, "w4"); print(w4);
    tensor* di = div(su, w4);       // (D, N)
    print(di);

    tensor* pw = pow(di, 2);
    tensor* ng = neg(pw);
    tensor* ex = exp(ng);
    tensor* out = log(ex);
    set_name(out, "out"); print(out);

    out->backward(out);
    graphviz(out);

    return 0;
}
