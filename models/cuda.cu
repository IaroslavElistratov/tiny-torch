#include <iostream> // todo: use C only
using namespace std;

#define DEVICE CUDA
#define print(a) ((DEVICE == CUDA) ? cuda_print_2d(a) : print_2d(a))

#include "../tensor.cpp"
#include "../ops.cpp"
#include "../utils.cpp"
#include "../print.cpp"


void cuda_print_2d(tensor* t)
{
    // todo: can just define a macro for print to call 4 lines below and then call the orignal print2d (no need for cuda_print_2d)
    hipDeviceSynchronize();
    int size = t->size * sizeof(float);
    float* host_data = (float*)malloc(size);
    hipError_t err = hipMemcpy(host_data, t->data, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        printf("[cuda memcopy] error: %s",  hipGetErrorString(err));
    }

    sprint_2d(t);

    int y = t->shape[0];
    int z = t->shape[1];

    for (int yi=0; yi<y; yi++){
        printf("[");
        for (int zi=0; zi<z; zi++){
            // todo: doens't make sense to use cpu strides when acessing contigious copy data
            // int idx = index_2d(t, yi, zi);
            int idx = yi * t->shape[1] + zi;
            printf("%8.4f, ", host_data[idx]);
        }
        printf("],\n");
    }
    printf("\n");
}


// todo-now:
// for cuda, use same operation abstractions (as for cpu), but make these abstractions call host stubs for cuda kernels (instead of my kernles for cpu) -- this will preserve my graph building functionality
//   - re-use this from ops.cpp: use same names for kernels, just make them refer to different impls (cpu, cuda) depending on wether device is CUDA or not -- this will reduce code duplication needed to copy paste ops

// todo-now:
// and "matmul_bwd", "batched_matmul_bwd", "div_bwd", "pow_bwd", "reduce_sum_bwd" can also be re-used!

int main() {
    // random num generator init, must be called once
    // srand(time(NULL));
    srand(123);

    int N = 16;
    int M = 8;
    int D = 4;

    tensor* x = CudaTensor(N, M);
    set_name(x, "x"); print(x);

    tensor* w1 = CudaTensor(M, D);
    set_name(w1, "w1"); print(w1);

    // x(N, M) @ w1(M, D) = out1(N, D)
    tensor* out = matmul(x, w1);
    set_name(out, "out"); print(out);



    x = CudaTensor(N, M);
    set_name(x, "x"); print(x);

    out = transpose(x);
    set_name(out, "out"); print(out);

    return 0;
}
